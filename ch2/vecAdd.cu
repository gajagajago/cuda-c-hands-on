#include <hip/hip_runtime.h>
#include <stdio.h>

#define threads_per_block 256.0

// n: vector size
__global__
void vecAddKernel(float* d_A, float* d_B, float* d_C, int n)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < n) {
        d_C[i] = d_A[i] + d_B[i];
    }
}

// n: vector size
void vecAdd(float* A, float* B, float* C, int n)
{
    // 1. Init device variables
    int size = n * sizeof(float);
    float *d_A, *d_B, *d_C;

    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    hipMemcpy(d_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, size, hipMemcpyHostToDevice);

    // 2. Compute
    dim3 dimGrid(ceil(n/threads_per_block), 1, 1);
    dim3 dimBlock(threads_per_block, 1, 1);
    vecAddKernel<<<dimGrid, dimBlock>>>(d_A, d_B, d_C, n);

    // 3. Device -> Host & Free device memory
    hipMemcpy(C, d_C, size, hipMemcpyDeviceToHost);

    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
}

int main(int argc, char* argv[])
{   
    int n = atoi(argv[1]);

    float A[n], B[n], C[n];

    for (int i=0; i<n; i++) {
        A[i] = B[i] = i;
    }

    vecAdd(A, B, C, n);

    for (int i=0; i<n; i++) {
        printf("%f\t", C[i]);
    }
    printf("\n");

    return 0;
}